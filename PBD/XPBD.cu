#include "XPBD.h"

namespace GPUPBD {
template <typename T>
XPBD<T>::XPBD(std::shared_ptr<Geometry<T>> geometry,T dt,int nRelax)
  :_geometry(geometry),_detector(new CollisionDetector<T>(geometry)),_dt(dt),_nRelax(nRelax) {}
template <typename T>
void XPBD<T>::step() {
  integrate();
  _detector->detectCollisions();
  initRelaxConstraint();
  for(int i=0; i<_nRelax; i++)
    relaxConstraint();
  updateVelocity();
}
template <typename T>
void XPBD<T>::integrate() {
  T dt = _dt;
  auto bIter = _geometry->getMutableCapsules().begin();
  auto eIter = _geometry->getMutableCapsules().end();
  thrust::for_each(thrust::device, bIter, eIter, [=] __host__ __device__ (Capsule<T>& capsule) {
    capsule._xPrev = capsule._x;
    capsule._v += capsule._force*dt/capsule._mass;
    capsule._x += capsule._v*dt;

    capsule._qPrev = capsule._q;
    capsule._R = capsule._q.toRotationMatrix();
    capsule._Iinv = capsule._R*capsule._Ibodyinv*capsule._R.transpose();
    capsule._w += capsule._Iinv*(capsule._torque
                                 - capsule._w.cross(capsule._R*capsule._Ibody*capsule._R.transpose()*capsule._w))*dt;
    Eigen::Quaternion<T> wQuat(0,capsule._w.x(),capsule._w.y(),capsule._w.z());
    Eigen::Quaternion<T> updatedQuat = Eigen::Quaternion<T>(0.5*dt,0,0,0)*wQuat*capsule._q;
    capsule._q = Eigen::Quaternion<T>(capsule._q.coeffs() + updatedQuat.coeffs());
    capsule._q.normalize();
  });
  hipDeviceSynchronize();
}
template <typename T>
void XPBD<T>::initRelaxConstraint() {
  if(_detector->size() == 0) {
    return;
  }
  _lambda.clear();
  _lambda.resize(_detector->size());
  _collisionCapsuleId.resize(_detector->size()*2); //each collision contains 2 capsules
  _deltaX.resize(_detector->size()*2);
  _deltaQ.resize(_detector->size()*2);
  _reduceCapsuleId.resize(_detector->size()*2);
  _reduceDeltaX.resize(_detector->size()*2);
  _reduceDeltaQ.resize(_detector->size()*2);

}
template <typename T>
void XPBD<T>::relaxConstraint() {
  if(_detector->size() == 0) {
    return;
  }
  const auto& collisions = _detector->getCollisions();
  const Collision<T>* d_collisions = thrust::raw_pointer_cast(collisions.data());
  auto& capsules = _geometry->getMutableCapsules();
  Capsule<T>* d_capsules = thrust::raw_pointer_cast(capsules.data());
  T* d_lambda = thrust::raw_pointer_cast(_lambda.data());
  int* d_collisionCapsuleId = thrust::raw_pointer_cast(_collisionCapsuleId.data());
  Vec3T* d_deltaX = thrust::raw_pointer_cast(_deltaX.data());
  Vec4T* d_deltaQ = thrust::raw_pointer_cast(_deltaQ.data());
  T dt = _dt;
  thrust::for_each(thrust::device,
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(static_cast<int>(collisions.size())),
  [=] __host__ __device__ (int idx) {
    auto& collision = d_collisions[idx];
    auto& cA = d_capsules[collision._capsuleIdA];
    auto& cB = d_capsules[collision._capsuleIdB];
    auto placementPointA = cA._q.toRotationMatrix()*collision._localPointA;
    auto placementPointB = cB._q.toRotationMatrix()*collision._localPointB;
    auto globalPointA = placementPointA+cA._x;
    auto globalPointB = placementPointB+cB._x;
    auto wA = computeGeneralizedInversMass(cA, placementPointA,
                                           collision._globalNormal);
    auto wB = computeGeneralizedInversMass(cB, placementPointB,
                                           collision._globalNormal);
    auto collisionDepth = (globalPointA-globalPointB).dot(collision._globalNormal);
    auto alpha = collision._alpha/(dt*dt);
    auto deltaLambda = (-collisionDepth-d_lambda[idx]*alpha)/(wA+wB+alpha);
    d_lambda[idx] += deltaLambda;
    auto pulse = deltaLambda*collision._globalNormal;
    // To avoid multi write problem, first cache update
    d_collisionCapsuleId[2*idx] = collision._capsuleIdA;
    d_collisionCapsuleId[2*idx+1] = collision._capsuleIdB;
    d_deltaX[2*idx] = pulse/cA._mass;
    d_deltaX[2*idx+1] = -pulse/cB._mass;
    d_deltaQ[2*idx] = getDeltaRot(cA, placementPointA, pulse).coeffs();
    d_deltaQ[2*idx+1] = -getDeltaRot(cB, placementPointB, pulse).coeffs();
  });

  updateCapsuleState();
}
template <typename T>
void XPBD<T>::updateVelocity() {
  T dt = _dt;
  auto bIter = _geometry->getMutableCapsules().begin();
  auto eIter = _geometry->getMutableCapsules().end();
  thrust::for_each(thrust::device, bIter, eIter, [=] __host__ __device__ (Capsule<T>& capsule) {
    capsule._v = (capsule._x-capsule._xPrev)/dt;
    auto deltaQ = capsule._q*capsule._qPrev.inverse();
    capsule._w = 2*deltaQ.vec()/dt;
    capsule._w = deltaQ.w() >=0 ? capsule._w : -capsule._w;
  });
  hipDeviceSynchronize();
}
template <typename T>
const CollisionDetector<T>& XPBD<T>::getDetector() const {
  if (!_detector) {
    throw std::runtime_error("Detector is not initialized");
  }
  return *_detector;
}
template <typename T>
DEVICE_HOST T XPBD<T>::computeGeneralizedInversMass(const Capsule<T>& c, const Vec3T& n, const Vec3T& r) {
  auto Iinv = c.getInertiaTensorInv();
  auto rCrossN = r.cross(n);
  auto w = 1.0/c._mass+rCrossN.transpose()*Iinv*rCrossN;
  return w;
}
template <typename T>
DEVICE_HOST Eigen::Quaternion<T> XPBD<T>::getDeltaRot(const Capsule<T>& c, const Vec3T& r, const Vec3T& pulse) {
  auto cIinv = c.getInertiaTensorInv();
  auto cIinvRCrossP = cIinv * (r.cross(pulse)); // I^{-1}(r x p)
  Eigen::Quaternion<T> cIinvRCrossPQuat(0,cIinvRCrossP.x(),cIinvRCrossP.y(),cIinvRCrossP.z());
  auto qUpdated = Eigen::Quaternion<T>(0.5,0,0,0)*cIinvRCrossPQuat*c._q;
  return qUpdated;
}
template <typename T>
void XPBD<T>::updateCapsuleState() {
  auto& capsules = _geometry->getMutableCapsules();
  Capsule<T>* d_capsules = thrust::raw_pointer_cast(capsules.data());
  //Reduce multi collisions of one capsule, then write
  auto endX = thrust::reduce_by_key(_collisionCapsuleId.begin(), _collisionCapsuleId.end(),
                                    _deltaX.begin(), _reduceCapsuleId.begin(), _reduceDeltaX.begin());
  _reduceCapsuleId.erase(endX.first, _reduceCapsuleId.end());
  int * d_reduceCapsuleId = thrust::raw_pointer_cast(_reduceCapsuleId.data());
  Vec3T* d_reduceDeltaX = thrust::raw_pointer_cast(_reduceDeltaX.data());
  thrust::for_each(thrust::device,
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(static_cast<int>(_reduceCapsuleId.size())),
  [=] __host__ __device__ (int idx) {
    d_capsules[d_reduceCapsuleId[idx]]._x = d_capsules[d_reduceCapsuleId[idx]]._x + d_reduceDeltaX[idx];
  });

  auto endQ = thrust::reduce_by_key(_collisionCapsuleId.begin(), _collisionCapsuleId.end(),
                                    _deltaQ.begin(), _reduceCapsuleId.begin(), _reduceDeltaQ.begin());
  _reduceCapsuleId.erase(endQ.first, _reduceCapsuleId.end());
  d_reduceCapsuleId = thrust::raw_pointer_cast(_reduceCapsuleId.data());
  Vec4T* d_reduceDeltaQ = thrust::raw_pointer_cast(_reduceDeltaQ.data());
  thrust::for_each(thrust::device,
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(static_cast<int>(_reduceCapsuleId.size())),
  [=] __host__ __device__ (int idx) {
    d_capsules[d_reduceCapsuleId[idx]]._q = Eigen::Quaternion<T>(d_capsules[d_reduceCapsuleId[idx]]._q.coeffs()
                                            + d_reduceDeltaQ[idx]);
    d_capsules[d_reduceCapsuleId[idx]]._q.normalize();
  });
}
//declare instance
template struct XPBD<LSCALAR>;
}
