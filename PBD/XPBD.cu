#include "XPBD.h"

namespace GPUPBD {
template <typename T>
XPBD<T>::XPBD(std::shared_ptr<Geometry<T>> geometry,T dt,int nRelax)
  :_geometry(geometry),_detector(new CollisionDetector<T>(geometry)),_dt(dt),_nRelax(nRelax) {}
template <typename T>
void XPBD<T>::step() {
  _detector->detectCollisions();
  integrate();
  for(int i=0; i<_nRelax; i++)
    relaxConstraint();
  updateVelocity();
}
template <typename T>
void XPBD<T>::integrate() {
  //to be implemeneted
}
template <typename T>
void XPBD<T>::relaxConstraint() {
  //to be implemeneted
}
template <typename T>
void XPBD<T>::updateVelocity() {
  //to be implemeneted
}
template <typename T>
const CollisionDetector<T>& XPBD<T>::getDetector() const {
  if (!_detector) {
    throw std::runtime_error("Detector is not initialized");
  }
  return *_detector;
}
//declare instance
template struct XPBD<LSCALAR>;
}
