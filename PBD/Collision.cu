#include "Collision.h"

namespace GPUPBD {
template <typename T>
CollisionDetector<T>::CollisionDetector(std::shared_ptr<Geometry<T>> geometry)
  :_geometry(geometry) {}
template <typename T>
void CollisionDetector<T>::detectCollisions() {
  FUNCTION_NOT_IMPLEMENTED
}
template <typename T>
Collision<T> CollisionDetector<T>::operator[](int id) {
  return _collisions[id];
}
template <typename T>
int CollisionDetector<T>::size() const {
  return _collisions.size();
}
}
