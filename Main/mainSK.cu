#include "hip/hip_runtime.h"
#include <PBD/Geometry.h>
#include <PBD/Collision.h>
#include <PBD/XPBD.h>
#include <PBD/Visualizer.h>
#include <TinyVisualizer/FirstPersonCameraManipulator.h>
#include <TinyVisualizer/CameraExportPlugin.h>
#include <TinyVisualizer/CaptureGIFPlugin.h>
#include <TinyVisualizer/ImGuiPlugin.h>
#include <TinyVisualizer/Camera3D.h>
#include <SKParser/tinyxml2.h>
#include <SKParser/Utils.h>

using namespace GPUPBD;

template <typename T>
struct Joint {
  DECL_MAT_VEC_MAP_TYPES_T
  bool _isValid=false;
  int _cA=-1;//son
  int _cB=-1;//parent
  Vec3T _cAPos;
  Vec3T _cBPos;
};
template <typename T>
struct Body {
  DECL_MAT_VEC_MAP_TYPES_T
  bool _isValid=false;
  int _parent;
  int _depth;
  T _radius;
  Vec3T _x;
  QuatT _q;
  Vec6T _ft;
  std::string _name;
  Shape<T> _c;
  Joint<T> _j;
};
template <typename T>
void readBodies(std::vector<Body<T>>& bodies, int parentId, const tinyxml2::XMLElement* g) {
  DECL_MAT_VEC_MAP_TYPES_T
  Body<T> body;
  body._parent=parentId;
  body._name=g->Attribute("name");
  //compute depth
  body._depth=parentId==-1?0:bodies[parentId]._depth+1;
  int _currId=(int)bodies.size();
  //read trans
  {
    body._x=PHYSICSMOTION::parsePtreeDef<Vec3T>(*g,"<xmlattr>.pos","0 0 0");
    Vec4T tmpQ=PHYSICSMOTION::parsePtreeDef<Vec4T>(*g,"<xmlattr>.quat","1 0 0 0");
    body._q=QuatT(tmpQ[0],tmpQ[1],tmpQ[2],tmpQ[3]);
  }
  //read geometry
  if(g->FirstChildElement("geom")->FindAttribute("type") == NULL) {
    //shape
    body._isValid=true;
    const tinyxml2::XMLElement* gg=g->FirstChildElement("geom");
    body._ft=PHYSICSMOTION::parsePtreeDef<Vec6T>(*gg,"<xmlattr>.fromto","0 0 0 0 0 0");
    body._radius=PHYSICSMOTION::get<T>(*gg,"<xmlattr>.size");
  } else {
    // TODO box
    body._isValid=false;
  }

  //read joints, temporarily ignore all angular constraints.
  if(g->FirstChildElement("joint")) {
    body._j._isValid=true;
    // parent:
    Body<T>& p = bodies[body._parent];
    Vec3T pC1 = Vec3T(p._ft[0], p._ft[1], p._ft[2]);
    Vec3T pC2 = Vec3T(p._ft[3], p._ft[4], p._ft[5]);
    Vec3T pX = (pC1+pC2)/2;
    QuatT pQ = QuatT::FromTwoVectors(Vec3T::UnitX(),(pC2-pC1).normalized());
    body._j._cB=body._parent;
    body._j._cBPos=pQ.inverse().toRotationMatrix()*(body._x-pX);
    // son:
    body._j._cA=_currId;
    Vec3T sC1 = Vec3T(body._ft[0], body._ft[1], body._ft[2]);
    Vec3T sC2 = Vec3T(body._ft[3], body._ft[4], body._ft[5]);
    Vec3T sX = (sC1+sC2)/2;
    QuatT sQ = QuatT::FromTwoVectors(Vec3T::UnitX(),(sC2-sC1).normalized());
    body._j._cAPos=-sQ.inverse().toRotationMatrix()*(sX);;
  } else {
    body._j._isValid=false;
  }

  if(!body._isValid) return;

  bodies.push_back(body);
  for(const tinyxml2::XMLElement* gc=g->FirstChildElement(); gc; gc=gc->NextSiblingElement())
    if(std::string(gc->Name()) == "body")
      readBodies(bodies,_currId,gc);
}

template <typename T>
void readMJCF(std::vector<Body<T>>& bodies, const std::string& file) {
  tinyxml2::XMLDocument pt;
  pt.LoadFile(file.c_str());
  tinyxml2::XMLElement* link=pt.RootElement();
  for(const tinyxml2::XMLElement* g=link->FirstChildElement(); g; g=g->NextSiblingElement())
    if(std::string(g->Name()) == "worldbody")
      readBodies(bodies,-1,g->FirstChildElement("body"));
}

template <typename T>
void updateShape(std::vector<Body<T>>& bodies) {
  DECL_MAT_VEC_MAP_TYPES_T
  for(auto& body : bodies) {
    Vec3T c1 = Vec3T(body._ft[0], body._ft[1], body._ft[2]);
    Vec3T c2 = Vec3T(body._ft[3], body._ft[4], body._ft[5]);
    c1 = body._x + body._q.toRotationMatrix() * c1;
    c2 = body._x + body._q.toRotationMatrix() * c2;
    if(body._parent>=0) {
      const auto& p = bodies[body._parent];
      c1 = p._x + p._q.toRotationMatrix()*c1;
      c2 = p._x + p._q.toRotationMatrix()*c2;
      body._x = p._x + p._q.toRotationMatrix()*body._x;
      body._q = (p._q * body._q).normalized();
    }
    body._c._radius=body._radius;
    body._c._len=(c2-c1).norm();
    body._c._x=(c1+c2)/2;
    body._c._q=QuatT::FromTwoVectors(Vec3T::UnitX(),(c2-c1).normalized());
  }
}


int main(int argc,char** argv) {
  typedef LSCALAR T;
  DECL_MAT_VEC_MAP_TYPES_T
  std::vector<Body<T>> bodies;
  readMJCF(bodies, "/data/GPU-PBD/SKParser/SK_Mannequin_PhysicsAsset_ABFB4_MJCF.xml");
  std::cout<<"==========================original info==========================" << std::endl;
  for(int i=0; i<bodies.size(); i++) {
    Body<T>& b = bodies[i];
    std::string indent(b._depth * 2, ' ');
    std::cout << indent  << b._name <<": x:" << b._x[0] << " " << b._x[1] << " " << b._x[2]
              << ", q:" << b._q.w() << " " << b._q.x() << " " << b._q.y() << " " << b._q.z()
              << ", radius:" << b._radius
              << ", fromto:" << b._ft[0] << " " << b._ft[1] << " "<< b._ft[2] << " "<< b._ft[3] << " "<< b._ft[4] << " "<< b._ft[5]
              << ", parent Name: " << bodies[b._parent>-1?b._parent:0]._name
              << ", joint._isValid: " << b._j._isValid
              << ", joint._cAName: " << bodies[b._j._cA>-1?b._j._cA:0]._name
              << ", joint._cBName: " << bodies[b._j._cB>-1?b._j._cB:0]._name
              << std::endl;
  }

  updateShape(bodies);
  std::cout<<"==========================updated info==========================" << std::endl;
  for(int i=0; i<bodies.size(); i++) {
    Body<T>& b = bodies[i];
    std::string indent(b._depth * 2, ' ');
    auto tmpx = b._c._q.toRotationMatrix() * Vec3T(b._c._len, 0, 0);
    std::cout << indent  << b._name <<": x:" << b._x[0] << " " << b._x[1] << " " << b._x[2]
              << ", q:" << b._q.w() << " " << b._q.x() << " " << b._q.y() << " " << b._q.z()
              << ", shape radius: " << b._c._radius << ", len:" << b._c._len
              <<", x:" << b._c._x[0] << " " << b._c._x[1] << " " << b._c._x[2]
              << ", q:" << b._c._q.w() << " " << b._c._q.x() << " " << b._c._q.y() << " " << b._c._q.z()
              << ", parent Name: " << bodies[b._parent>-1?b._parent:0]._name
              << std::endl;
  }

  std::vector<Shape<T>> ps;
  for(auto& b : bodies) {
    Shape<T> c = b._c;
    c._type = ShapeType::Capsule;
    c._v.setZero();
    c._w.setZero();
    c._torque.setZero();
    c.initInertiaTensor();
    c._force = Vec3T(0, -9.8f*c._mass,0);
    c._isDynamic = true;
    ps.push_back(c);
  }

  // boundary
  Shape<T> b_1;
  b_1._type = ShapeType::Capsule;
  b_1._len = 20;
  b_1._radius = 1;
  b_1._mass = 1;
  b_1._x = Vec3T(0,-4,0);
  b_1._q = QuatT(1,0,0,0);
  b_1.initInertiaTensor();
  b_1._isDynamic = false;
  ps.push_back(b_1);

  Shape<T> b_2;
  b_2._type = ShapeType::Capsule;
  b_2._len = 20;
  b_2._radius = 1;
  b_2._mass = 1;
  b_2._x = Vec3T(0,-4,1);
  b_2._q = QuatT(1,0,0,0);
  b_2.initInertiaTensor();
  b_2._isDynamic = false;
  ps.push_back(b_2);

  Shape<T> b_3;
  b_3._type = ShapeType::Capsule;
  b_3._len = 20;
  b_3._radius = 1;
  b_3._mass = 1;
  b_3._x = Vec3T(0,-4,-1);
  b_3._q = QuatT(1,0,0,0);
  b_3.initInertiaTensor();
  b_3._isDynamic = false;
  ps.push_back(b_3);

  Shape<T> b_4;
  b_4._type = ShapeType::Capsule;
  b_4._len = 20;
  b_4._radius = 1;
  b_4._mass = 1;
  b_4._x = Vec3T(0,-4,2);
  b_4._q = QuatT(1,0,0,0);
  b_4.initInertiaTensor();
  b_4._isDynamic = false;
  ps.push_back(b_4);

  Shape<T> b_5;
  b_5._type = ShapeType::Capsule;
  b_5._len = 20;
  b_5._radius = 1;
  b_5._mass = 1;
  b_5._x = Vec3T(0,-4,-2);
  b_5._q = QuatT(1,0,0,0);
  b_5.initInertiaTensor();
  b_5._isDynamic = false;
  ps.push_back(b_5);

  std::shared_ptr<Geometry<T>> geometry(new Geometry<T>);
  geometry->resize(ps.size());
  geometry->setShape(ps);
  XPBD<T> xpbd(geometry, 1.0f/60);
  // addJoint
  std::cout<<"==========================joint info==========================" << std::endl;
  for(auto& b : bodies) {
    auto& j = b._j;
    if(j._isValid) {
      xpbd.addJoint(j._cA,j._cB,j._cAPos,j._cBPos);
      std::cout<<"Parent: " << bodies[j._cB]._name << ", Self: " << bodies[j._cA]._name
               << ", Parent Pos: " << j._cBPos[0] << ", "  << j._cBPos[2] << ", " << j._cBPos[2]
               << ", Self Pos: " << j._cAPos[0] << ", "  << j._cAPos[2] << ", " << j._cAPos[2]
               << std::endl;
    }
  }
  DRAWER::Drawer drawer(argc,argv);
  drawer.addPlugin(std::shared_ptr<DRAWER::Plugin>(new DRAWER::CameraExportPlugin(GLFW_KEY_2,GLFW_KEY_3,"camera.dat")));
  drawer.addPlugin(std::shared_ptr<DRAWER::Plugin>(new DRAWER::CaptureGIFPlugin(GLFW_KEY_1,"record.gif",drawer.FPS())));
  auto shapeGeometry=visualizeOrUpdateGeometry(*geometry);
  auto shapeCollision=visualizeOrUpdateCollision(*geometry,xpbd.getDetector());
  drawer.addShape(shapeGeometry);
  drawer.addShape(shapeCollision);
  drawer.addCamera3D(90,Eigen::Matrix<GLfloat,3,1>(0,1,0),Eigen::Matrix<GLfloat,3,1>(0,0,5),Eigen::Matrix<GLfloat,3,1>(0,0,-1));
  drawer.getCamera3D()->setManipulator(std::shared_ptr<DRAWER::CameraManipulator>(new DRAWER::FirstPersonCameraManipulator(drawer.getCamera3D())));
  drawer.addPlugin(std::shared_ptr<DRAWER::Plugin>(new DRAWER::ImGuiPlugin([&]() {
    drawer.getCamera3D()->getManipulator()->imGuiCallback();
  })));
  bool sim=false;
  drawer.setFrameFunc([&](std::shared_ptr<DRAWER::SceneNode>& root) {
    if(sim) {
      xpbd.step();
      visualizeOrUpdateGeometry(*geometry,shapeGeometry);
      visualizeOrUpdateCollision(*geometry,xpbd.getDetector(),shapeCollision);
    }
  });
  //press R to run simulation
  drawer.setKeyFunc([&](GLFWwindow* wnd,int key,int scan,int action,int mods,bool captured) {
    if(captured)
      return;
    else if(key==GLFW_KEY_R && action==GLFW_PRESS)
      sim=!sim;
  });
  drawer.mainLoop();

  return 0;
}
